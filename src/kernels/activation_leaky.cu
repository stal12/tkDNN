#include "hip/hip_runtime.h"
#include "tkDNN/kernels.h"

__global__
void activation_leaky(dnnType *input, dnnType *output, int size, float slope) {

    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i<size) {    
        if (input[i]>0)
            output[i] = input[i];
        else
            output[i] = slope*input[i];
    }
 }


/**
    ELU activation function
*/
void activationLEAKYForward(dnnType* srcData, dnnType* dstData, int size, float slope, hipStream_t stream)
{
    int blocks = (size+255)/256;
    int threads = 256;
    
    activation_leaky<<<blocks, threads, 0, stream>>>(srcData, dstData, size, slope);
}


